#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

struct BasicLight
{
 optix::float3 pos;
 optix::float3 color;
 int casts_shadow;
 };

 struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};
 

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, color_normal, attribute color_normal,);

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(float3,   Ka, , ); 
rtDeclareVariable(float3,   Ks, , ); 
rtDeclareVariable(float,    phong_exp, , );
rtDeclareVariable(float3,   Kd, , ); 
rtDeclareVariable(float3,   ambient_light_color, , );
rtBuffer<BasicLight>        lights;
rtDeclareVariable(rtObject, top_shadower, , );






RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = make_float3(0.0f);
  
	rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;

	for(int i = 0; i <lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot( ffnormal, L);

		if( nDl >= 0.0f ){
			// cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if( fmaxf(light_attenuation) > 0.0f ){
				float3 Lc = Kd*color_normal;// * light_attenuation;
				color += Lc;
			}

		}
	}
	prd_radiance.result = Ks*color_normal+color/lights.size();
}


//normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));