#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, sphere, , );
rtDeclareVariable(float3, sphere_color, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx)
{
	const float3 center = make_float3(sphere);
	const float3 O = ray.origin - center;
	const float3 D = ray.direction;
	const float radius = sphere.w;

	float b = dot(O, D);
	float c = dot(O, O) - radius * radius;
	float disc = b * b - c;

	if (disc <= 0.0f) return;

	float sdisc = sqrtf(disc);
	float root1 = (-b - sdisc);

	if (rtPotentialIntersection(root1))
	{
		shading_normal = geometric_normal = sphere_color;
		rtReportIntersection(0);
	} 
}

RT_PROGRAM void bounds(int, float result[6])
{
	const float3 center = make_float3(sphere);
	const float radius = sphere.w;

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (radius <= 0.0f || isinf(radius))
	{
		aabb->invalidate();
		return;
	}

	aabb->m_min = center - make_float3(radius);
	aabb->m_max = center + make_float3(radius);
}

