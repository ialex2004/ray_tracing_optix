#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

struct BasicLight
{
 optix::float3 pos;
 optix::float3 color;
 int casts_shadow;
 };

rtBuffer<float4> spheres;
rtBuffer<float3> spheres_colors;


rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, color_normal, attribute color_normal,);
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx)
{
	int num_spheres=spheres.size();
	int i;
	
	#pragma unroll
	for (i=0; i<num_spheres;i++)
	{
		const float3 center = make_float3(spheres[i]);
	
		const float3 O = ray.origin - center;
		const float3 D = ray.direction;
		const float radius = spheres[i].w;

		float b = dot(O, D);
		float c = dot(O, O) - radius * radius;
		float disc = b * b - c;

		if (!(disc <= 0.0f))
		{
			float sdisc = sqrtf(disc);
			float root1 = (-b - sdisc);
	
			if (rtPotentialIntersection(root1))
			{
					
				shading_normal = geometric_normal = (O + (root1 )*D)/radius;
				color_normal=spheres_colors[i];
				rtReportIntersection(0);
			}
		}
	}
	return;
}

RT_PROGRAM void bounds(int, float result[6])
{
	int num_spheres=spheres.size();
	int i;
	#pragma unroll
	for (i=0; i<num_spheres;i++)
	{
		const float3 center = make_float3(spheres[i]);
		
		const float radius = spheres[i].w;

		optix::Aabb* aabb = (optix::Aabb*)result;

		if (radius <= 0.0f || isinf(radius))
		{
			aabb->invalidate();
			
		}
		else 
		{
		aabb->m_min = center - make_float3(radius);
		aabb->m_max = center + make_float3(radius);
		}
	}
	return;
}

